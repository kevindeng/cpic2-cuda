#include "hip/hip_runtime.h"
extern "C"
{
    #include <stdlib.h>
    #include <stdio.h>
    #include <complex.h>
    #include <math.h>
    #include "push2cuda.h"
}

#define HANDLE_ERROR(A,M) {hipError_t e;e=A;if(e!=hipSuccess){printf("CUDA error: %s\n",M);exit(1);}}

int floatArrayCompare(float* a, float* b, int n, char* name1, char* name2, float epsilon)
{
    int failed = 0;
    for(int i = 0; i < n; i++)
    {
        float diff = fabs(a[i] - b[i]);
        if(diff > epsilon)
        {
            printf("%d: %s: %.7e %s: %.7e diff: %.7e\n", i, name1, name2, a[i], b[i], diff); 
            failed = 1;
        }
    }
    return failed;
}

void* copyToGPU(void* a, int n)
{
    void* g;
    HANDLE_ERROR(hipMalloc((void**)&g, n), "hipMalloc in copyToGPU");
    HANDLE_ERROR(hipMemcpy(g, a, n, hipMemcpyHostToDevice), "hipMemcpy in copyToGPU");
    return g;
}

void* copyFromGPU(void* g, int n)
{
    void* a = malloc(n);
    if(!a)
    {
        printf("malloc failed.");
        exit(1);
    }
    HANDLE_ERROR(hipMemcpy(a, g, n, hipMemcpyDeviceToHost), "hipMemcpy in copyFromGPU");
    return a;
}

void copyToGPU2(void* g, void* h, int n)
{
    hipMemcpy(g, h, n, hipMemcpyHostToDevice);
}

void copyFromGPU2(void* h, void* g, int n)
{
    hipMemcpy(h, g, n, hipMemcpyDeviceToHost);
}

void freeOnGPU(void* g)
{
    hipFree(g);
}