#include "hip/hip_runtime.h"
extern "C"
{
    #include <stdlib.h>
    #include <stdio.h>
    #include <complex.h>
    #include <math.h>
    #include "push2cuda.h"
}

#define HANDLE_ERROR(A,M) {hipError_t e;e=A;if(e!=hipSuccess){printf("CUDA error: %s\n",M);exit(1);}}

int floatArrayCompare(float* a, float* b, int n, char* name1, char* name2, float epsilon)
{
    int failed = 0;
    for(int i = 0; i < n; i++)
    {
        float diff = fabs(a[i] - b[i]);
        if(diff > epsilon)
        {
            printf("%d: %s: %.7e %s: %.7e diff: %.7e\n", i, name1, name2, a[i], b[i], diff); 
            failed++;
        }
    }
    if(failed)
        printf("%d of %d failed\n", failed, n);
    return failed;
}

void* copyToGPU(void* a, int n)
{
    void* g;
    HANDLE_ERROR(hipMalloc((void**)&g, n), "hipMalloc in copyToGPU");
    HANDLE_ERROR(hipMemcpy(g, a, n, hipMemcpyHostToDevice), "hipMemcpy in copyToGPU");
    return g;
}

void* copyFromGPU(void* g, int n)
{
    void* a = malloc(n);
    if(!a)
    {
        printf("malloc failed.");
        exit(1);
    }
    HANDLE_ERROR(hipMemcpy(a, g, n, hipMemcpyDeviceToHost), "hipMemcpy in copyFromGPU");
    return a;
}

void copyToGPU2(void* g, void* h, int n)
{
    HANDLE_ERROR(hipMemcpy(g, h, n, hipMemcpyHostToDevice), "hipMemcpy in copyToGPU2");
}

void copyFromGPU2(void* h, void* g, int n)
{
    HANDLE_ERROR(hipMemcpy(h, g, n, hipMemcpyDeviceToHost), "hipMemcpy in copyFromGPU2");
}

void freeOnGPU(void* g)
{
    hipFree(g);
}

int* createMutexes(int n)
{
    int* mut;
    HANDLE_ERROR(hipMalloc((void**)&mut, n * sizeof(int)), "hipMalloc in createMutexes");
    hipMemset(mut, 0, n * sizeof(int));
    return mut;
}

__device__ void syncAddFloat(float* target, float amt, int* mutex)
{
    while(atomicCAS(mutex, 0, 1) != 0);
    (*target) += amt;
    atomicExch(mutex, 0);
}

/*---------------------------------------------------------------------------*/
/*--------------------------------- cgpost2l --------------------------------*/
/*---------------------------------------------------------------------------*/

__global__ void k_cgpost2l(float* part, float* q, float qm, int nop, int idimp,
            int nxv, int nyv, int* mutexes)
{
    int j, nn, mm, np, mp;
    float dxp, dyp, amx, amy;
    /* find interpolation weights */
    j = blockIdx.x + blockIdx.y * gridDim.x;
    if(j < nop) 
    //for(j = 0; j < nop; j++)
    {
        nn = part[idimp*j];
        mm = part[1+idimp*j];
        dxp = qm*(part[idimp*j] - (float) nn);
        dyp = part[1+idimp*j] - (float) mm;
        mm = nxv*mm;
        amx = qm - dxp;
        mp = mm + nxv;
        amy = 1.0 - dyp;
        np = nn + 1;
    /* deposit charge */
        syncAddFloat(q+np+mp, dxp*dyp, mutexes+np+mp);
        syncAddFloat(q+nn+mp, amx*dyp, mutexes+nn+mp);
        syncAddFloat(q+np+mm, dxp*amy, mutexes+np+mm);
        syncAddFloat(q+nn+mm, amx*amy, mutexes+nn+mm);
        /*q[np+mp] += dxp*dyp;
        q[nn+mp] += amx*dyp;
        q[np+mm] += dxp*amy;
        q[nn+mm] += amx*amy;*/
    }
}

void cgpost2l_cuda(float* part, float* q, float qm, int nop, int idimp,
            int nxv, int nyv, int npx, int npy, int* mutexes) 
{
    hipMemset(q, 0, nxv * nyv * sizeof(float));
    dim3 grid(npx, npy);
    k_cgpost2l<<<grid, 1>>>(part, q, qm, nop, idimp, nxv, nyv, mutexes);
}



/*---------------------------------------------------------------------------*/
/*-------------------------------- caguard2l --------------------------------*/
/*---------------------------------------------------------------------------*/

__global__ void k_caguard2l(float* q, int nx, int ny, int nxe, int nye)
{
    int j, k;
    /* accumulate edges of extended field */
    j = blockIdx.x - ny;
    k = blockIdx.x;
    if(k < ny)
    {
        q[nxe*k] += q[nx+nxe*k];
        q[nx+nxe*k] = 0.0;
    }
    if(j >= 0 && j < nx)
    {
        q[j] += q[j+nxe*ny];
        q[j+nxe*ny] = 0.0;
    }
    if(k == 0)
    {
        q[0] += q[nx+nxe*ny];
        q[nx+nxe*ny] = 0.0;
    }
}

void caguard2l_cuda(float* q, int nx, int ny, int nxe, int nye) 
{
    k_caguard2l<<<nx + ny, 1>>>(q, nx, ny, nxe, nye);
}



/*---------------------------------------------------------------------------*/
/*-------------------------------- cgpush2l --------------------------------*/
/*---------------------------------------------------------------------------*/
__global__ void k_cgpush2l(float* part, float* fxy, float qbm, float dt, float *ek,
              int idimp, int nop, int nx, int ny, int nxv, int nyv,
              int ipbc, int* mutex)
{
    int j, nn, mm, np, mp, nxv2;
    float qtm, edgelx, edgely, edgerx, edgery, dxp, dyp, amx, amy;
    float dx, dy, vx, vy;
    double sum1;
    nxv2 = 2*nxv;
    qtm = qbm*dt;
    sum1 = 0.0;
    /* set boundary values */
    edgelx = 0.0;
    edgely = 0.0;
    edgerx = (float) nx;
    edgery = (float) ny;
    if (ipbc==2) {
        edgelx = 1.0;
        edgely = 1.0;
        edgerx = (float) (nx-1);
        edgery = (float) (ny-1);
    }
    else if (ipbc==3) {
        edgelx = 1.0;
        edgerx = (float) (nx-1);
    }

    j = blockIdx.x + blockIdx.y * gridDim.x;
    if(j < nop)
    {
    //for (j = 0; j < nop; j++) {
    /* find interpolation weights */
        nn = part[idimp*j];
        mm = part[1+idimp*j];
        dxp = part[idimp*j] - (float) nn;
        dyp = part[1+idimp*j] - (float) mm;
        nn = 2*nn;
        mm = nxv2*mm;
        amx = 1.0 - dxp;
        mp = mm + nxv2;
        amy = 1.0 - dyp;
        np = nn + 2;
    /* find acceleration */
        dx = dyp*(dxp*fxy[np+mp] + amx*fxy[nn+mp])
            + amy*(dxp*fxy[np+mm] + amx*fxy[nn+mm]);
        dy = dyp*(dxp*fxy[1+np+mp] + amx*fxy[1+nn+mp])
            + amy*(dxp*fxy[1+np+mm] + amx*fxy[1+nn+mm]);
    /* new velocity */
        vx = part[2+idimp*j];
        vy = part[3+idimp*j];
        dx = vx + qtm*dx;
        dy = vy + qtm*dy;
    /* average kinetic energy */
        vx += dx;
        vy += dy;
        sum1 += vx*vx + vy*vy;
        part[2+idimp*j] = dx;
        part[3+idimp*j] = dy;
    /* new position */
        dx = part[idimp*j] + dx*dt;
        dy = part[1+idimp*j] + dy*dt;
    /* periodic boundary conditions */
        if (ipbc==1) {
            if (dx < edgelx) dx += edgerx;
            if (dx >= edgerx) dx -= edgerx;
            if (dy < edgely) dy += edgery;
            if (dy >= edgery) dy -= edgery;
        }
    /* reflecting boundary conditions */
        else if (ipbc==2) {
            if ((dx < edgelx) || (dx >= edgerx)) {
                dx = part[idimp*j];
                part[2+idimp*j] = -part[2+idimp*j];
            }
            if ((dy < edgely) || (dy >= edgery)) {
                dy = part[1+idimp*j];
                part[3+idimp*j] = -part[3+idimp*j];
            }
        }
    /* mixed reflecting/periodic boundary conditions */
        else if (ipbc==3) {
            if ((dx < edgelx) || (dx >= edgerx)) {
                dx = part[idimp*j];
                part[2+idimp*j] = -part[2+idimp*j];
            }
            if (dy < edgely) dy += edgery;
            if (dy >= edgery) dy -= edgery;
        }
    /* set new position */
        part[idimp*j] = dx;
        part[1+idimp*j] = dy;
    }
    /* normalize kinetic energy */
    syncAddFloat(ek, .125*sum1, mutex);
}

void cgpush2l_cuda(float part[], float fxy[], float qbm, float dt, float *ek,
              int idimp, int nop, int nx, int ny, int nxv, int nyv,
              int ipbc, int npx, int npy, int* mutex) 
{                
    dim3 grid(npx, npy);
    k_cgpush2l<<<grid, 1>>>(part, fxy, qbm, dt, ek, idimp, nop, nx, ny, nxv, nyv, ipbc, mutex);
}